#include "hip/hip_runtime.h"
#include "gpu_bc_node.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define THREAD_NUM 256
#define DEBUG

__global__ void forward_kernel (int *outgoing_starts, int *outgoing_edges, 
    int *d, int *sigma, bool *cont, int *dist, int num_nodes) {
	int v = blockIdx.x * blockDim.x + threadIdx.x;

  if(v >= num_nodes){
    return;
  }

  if(d[v] == *dist) {
    int start = outgoing_starts[v];
    int end = outgoing_starts[v + 1];
    for(int p = start; p < end; p++) {
      int w = outgoing_edges[p];
      if(d[w] == NOT_VISITED_MARKER) {
        d[w] = *dist + 1;
        *cont = true;
      }
      if(d[w] == *dist + 1) {
        atomicAdd(&sigma[w], sigma[v]);
      }
    }
  }
}

__global__ void backward_kernel (int *outgoing_starts, 
    int* outgoing_edges, int *d, int *sigma, float *delta, float* bc, 
    int *dist, int num_nodes) {
  int v = blockIdx.x * blockDim.x + threadIdx.x;

  if(v >= num_nodes) {
    return;
  }

  if(d[v] == *dist - 1) {
    int start = outgoing_starts[v];
    int end = outgoing_starts[v + 1];
    float sum = 0;
    for(int p = start; p < end; p++) {
      int w = outgoing_edges[p];
      if(d[w] == *dist) {
        sum += (float)sigma[v] / sigma[w] * (delta[w] + 1);
      }
    }
    delta[v] += sum;
  }
}

__global__ void compute_bc_kernel (int node_id, int *d, float *delta, 
    float *bc, int num_nodes) {
  int v = blockIdx.x * blockDim.x + threadIdx.x;

  if(v < num_nodes && v != node_id && d[v] != NOT_VISITED_MARKER) {
    bc[v] += delta[v];
  }
}

/*
__global__ void compute_bc_kernel_deg1 (int s, int *d, float *delta, float *bc, int num_nodes, int* d_weight) {
	int v = blockIdx.x * blockDim.x + threadIdx.x;
	if(v < num_nodes && v != s && d[v] != -1) {
		bc[v] += delta[v] * d_weight[s];
	}
}
*/

__global__ void init_params_kernel (int s, int *d, int *sigma, 
    int num_nodes, int* dist){
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if(i >= num_nodes) {
    return;
  }

  if(s == i) {
    d[i] = 0;
    sigma[i] = 1;
    *dist = 0;
  } else {
    d[i] = -1;
    sigma[i] = 0;
  }
}

/*
__global__ void set_int_vertex (int* dest, int val){
	*dest = val;
}
*/

/*
__global__ void init_delta (int *d_weight, float* delta, int num_nodes) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < num_nodes) {
		delta[i] = d_weight[i]-1;
	}
}
*/

void setup(const graph *g, int **outgoing_starts, int **outgoing_edges, 
    int **d, int **sigma, float **delta, int **dist, float **bc, 
    bool **d_cont) {
  int num_nodes = g->num_nodes;
  int num_edges = g->num_edges;

  hipMalloc((void **)outgoing_starts, sizeof(int) * (num_nodes + 1));
  hipMalloc((void **)outgoing_edges, sizeof(int) * num_edges);

  hipMemcpy(*outgoing_starts, g->outgoing_starts, 
      sizeof(int) * (num_nodes + 1), hipMemcpyHostToDevice); 
  hipMemcpy(*outgoing_edges, g->outgoing_edges, 
      sizeof(int) * num_edges, hipMemcpyHostToDevice); 

  hipMalloc((void **)d, sizeof(int) * num_nodes);

  hipMalloc((void **)sigma, sizeof(int) * num_nodes);
  hipMalloc((void **)delta, sizeof(float) * num_nodes);
  hipMalloc((void **)dist, sizeof(int));

  hipMalloc((void **)bc, sizeof(float) * num_nodes);
  hipMemset(*bc, 0, sizeof(float) * num_nodes);

  hipMalloc((void **)d_cont, sizeof(bool));
}

void clean(int **outgoing_starts, int **outgoing_edges, int **d, 
    int **sigma, float **delta, int **dist, float **bc, bool **d_cont) {
  hipFree(outgoing_starts);
  hipFree(outgoing_edges);
  hipFree(d);
  hipFree(sigma);
  hipFree(delta);
  hipFree(dist);
  hipFree(bc);
  hipFree(d_cont);
}

int gpu_bc_node (const graph *g, float *bc) {
  int *device_outgoing_starts, *device_outgoing_edges;
  int *device_d, *device_sigma, *device_dist, distance;
  float *device_delta, *device_bc;
  bool cont, *device_cont;
  int num_nodes = g->num_nodes;

  setup(g, &device_outgoing_starts, &device_outgoing_edges, &device_d, 
      &device_sigma, &device_delta, &device_dist, &device_bc, &device_cont);

  dim3 blockDim(THREAD_NUM);
  dim3 gridDim((g->num_nodes + blockDim.x - 1) / blockDim.x);

  for(int node_id = 0; node_id < num_nodes; node_id++) {
    distance = 0;
    init_params_kernel<<<gridDim,blockDim>>>(node_id, device_d, device_sigma, 
      num_nodes, device_dist);

    // BFS
    do {
      hipMemset(device_cont, false, sizeof(bool));
      forward_kernel<<<gridDim, blockDim>>>(device_outgoing_starts, 
        device_outgoing_edges, device_d, device_sigma, device_cont, 
        device_dist, num_nodes);
      hipDeviceSynchronize();
      hipMemcpy(device_dist, &(++distance), sizeof(int), hipMemcpyHostToDevice);
      hipMemcpy(&cont, device_cont, sizeof(bool), hipMemcpyDeviceToHost);
    } while (cont);


    //Back propagation
    hipMemset(device_delta, 0, sizeof(int) * num_nodes);
    hipMemcpy(device_dist, &(--distance), sizeof(int), hipMemcpyHostToDevice);
    while (distance > 1) {
      backward_kernel<<<gridDim, blockDim>>>(device_outgoing_starts, 
        device_outgoing_edges, device_d, device_sigma, device_delta, 
        device_bc, device_dist, num_nodes);
      hipDeviceSynchronize();
      hipMemcpy(device_dist, &(--distance), sizeof(int), hipMemcpyHostToDevice);
    }
    compute_bc_kernel<<<gridDim, blockDim>>>(node_id, device_d, 
      device_delta, device_bc, num_nodes);
  }

  hipMemcpy(bc, device_bc, sizeof(float)*num_nodes, hipMemcpyDeviceToHost);

  clean(&device_outgoing_starts, &device_outgoing_edges, &device_d, 
      &device_sigma, &device_delta, &device_dist, &device_bc, &device_cont);

  return 0;
}
/*
int bc_vertex_deg1 (int *h_ptrs, int* h_js, int num_nodes, int num_edges, int nb, float *bc, int* h_weight) {

	int *device_outgoing_starts, *device_outgoing_edges, *device_d, *device_sigma, *device_dist, distance, *d_weight;
	float *device_delta, *device_bc;
	bool h_continue, *device_continue;

	hipMalloc((void **)&device_outgoing_starts, sizeof(int) * (num_nodes + 1));
	hipMalloc((void **)&device_outgoing_edges, sizeof(int) * num_edges);

	hipMemcpy(device_outgoing_starts, h_ptrs, sizeof(int) * (num_nodes+1), hipMemcpyHostToDevice); // xadj array
	hipMemcpy(device_outgoing_edges, h_js, sizeof(int) * num_edges, hipMemcpyHostToDevice); // adj array

	hipMalloc((void **)&device_d, sizeof(int) * num_nodes);

	hipMalloc((void **)&device_sigma, sizeof(int) * num_nodes);
	hipMalloc((void **)&device_delta, sizeof(float) * num_nodes);
	hipMalloc((void **)&d_weight, sizeof(int) * num_nodes);
	hipMemcpy(d_weight, h_weight, sizeof(int) * num_nodes, hipMemcpyHostToDevice); // weight array
	hipMalloc((void **)&device_dist, sizeof(int));

	hipMalloc((void **)&device_bc, sizeof(float) * num_nodes);
	hipMemcpy(device_bc, bc, sizeof(int) * num_nodes, hipMemcpyHostToDevice); // bc array

	hipMalloc((void **)&device_continue, sizeof(bool));

	int threads_per_block = num_nodes;
	int blocks = 1;
	if(num_nodes > MTS){
		blocks = (int)ceil(num_nodes/(double)MTS);
		threads_per_block = MTS;
	}

	dim3 grid(blocks);
	dim3 threads(threads_per_block);


#ifdef TIMER
	struct timeval t1, t2, gt1, gt2; double time;
#endif

	for(int i = 0; i < min (nb, num_nodes); i++){
#ifdef TIMER
		gettimeofday(&t1, 0);
#endif

		distance = 0;
		init_params_kernel<<<grid,threads>>>(i, device_d, device_sigma, num_nodes, device_dist);

#ifdef TIMER
		gettimeofday(&t2, 0);
		time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0;
		cout << "initialization takes " << time << " secs\n";
		gettimeofday(&gt1, 0);
#endif

		// BFS
		do{
#ifdef TIMER
			gettimeofday(&t1, 0);
#endif

			hipMemset(device_continue, 0, sizeof(bool));
			forward_kernel<<<grid,threads>>>(device_outgoing_starts, device_outgoing_edges, device_d, device_sigma, device_continue, device_dist, num_nodes);
			hipDeviceSynchronize();
			set_int_vertex<<<1,1>>>(device_dist, ++distance);
			hipMemcpy(&h_continue, device_continue, sizeof(bool), hipMemcpyDeviceToHost);

#ifdef TIMER
			gettimeofday(&t2, 0);
			time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0;
			cout << "level " << distance << " takes " << time << " secs\n";
#endif

		} while(h_continue);

#ifdef TIMER
		gettimeofday(&gt2, 0);
		time = (1000000.0*(gt2.tv_sec-gt1.tv_sec) + gt2.tv_usec-gt1.tv_usec)/1000000.0;
		cout << "Phase 1 takes " << time << " secs\n";
		gettimeofday(&gt1, 0); // starts back propagation
#endif

		//Back propagation

		init_delta<<<grid, threads>>>(d_weight, device_delta, num_nodes); // deltas are initialized
		set_int_vertex<<<1,1>>>(device_dist, --distance);
		while(distance > 1) {
			backward_kernel<<<grid, threads>>>(device_outgoing_starts, device_outgoing_edges, device_d, device_sigma, device_delta, device_bc, device_dist, num_nodes);
			hipDeviceSynchronize();
			set_int_vertex<<<1,1>>>(device_dist, --distance);
		}


		compute_bc_kernel_deg1<<<grid, threads>>>(i, device_d, device_delta, device_bc, num_nodes, d_weight);
		hipDeviceSynchronize();

#ifdef TIMER
		gettimeofday(&gt2, 0);
		time = (1000000.0*(gt2.tv_sec-gt1.tv_sec) + gt2.tv_usec-gt1.tv_usec)/1000000.0;
		cout << "Phase 2 takes " << time << " secs\n";
#endif

	}

	hipMemcpy(bc, device_bc, sizeof(float)*num_nodes, hipMemcpyDeviceToHost);
	hipFree(device_outgoing_starts);
	hipFree(device_outgoing_edges);
	hipFree(device_d);
	hipFree(device_sigma);
	hipFree(device_delta);
	hipFree(device_dist);
	hipFree(device_bc);
	hipFree(device_continue);


	return 0;
}
*/
