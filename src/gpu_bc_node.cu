#include "hip/hip_runtime.h"
#include "graph.h"
#include "gpu_bc_node.h"
#include "CycleTimer.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define DEBUG

const int NOT_VISITED_MARKER = -1;

__global__ void init_params(int num_nodes, int *d, int *sigma, int node_id) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= num_nodes) {
    return;
  }

  if (i != node_id) {
    d[i] = NOT_VISITED_MARKER;
    sigma[i] = 0;
  } else {
    d[i] = 0;
    sigma[i] = 1;
  }
}

__global__ void forward_propagation_kernel(int *outgoing_starts, 
  int *outgoing_edges, int num_nodes, int num_edges, int *d, 
  int *sigma, int *distance, bool *done) {
  int v = blockIdx.x * blockDim.x + threadIdx.x;

  if (v >= num_nodes) {
    return;
  }

  if (d[v] == *distance) {
    int start_edge = outgoing_starts[v];
    int end_edge = (v == num_nodes - 1)?  num_edges: outgoing_starts[v + 1];
#ifdef DEBUG
    printf("(start, end): (%d, %d)\n", start_edge, end_edge);
#endif
    for (int neighbor = start_edge; neighbor < end_edge; ++neighbor) {
      int w = outgoing_edges[neighbor];
      
      if (d[w] == NOT_VISITED_MARKER) {
#ifdef DEBUG
        printf("set done here\n");
#endif
        d[w] = *distance + 1;
        *done = false;
      }
      if (d[w] == *distance + 1) {
        atomicAdd(&sigma[w], sigma[v]);
      }
    }
  }
}

__global__ void backward_propagation_kernel(int *outgoing_starts,
  int *outgoing_edges, int num_nodes, int num_edges, int *d, 
  int *sigma, float *delta, int *distance) {
  int v = blockIdx.x * blockDim.x + threadIdx.x;

  if (v >= num_nodes) {
    return;
  }

  if (d[v] == *distance) {
    int start_edge = outgoing_starts[v];
    int end_edge = (v == num_nodes - 1)? 
      num_edges: outgoing_starts[v + 1];
    float sum = 0;

    // loop through all neighbors
    for (int neighbor = start_edge; neighbor < end_edge; ++neighbor) {
      int w = outgoing_edges[neighbor];
      if (d[w] == *distance + 1) {
        sum += (float)sigma[v] / sigma[w] * (delta[w] + 1);
      }
    }
    delta[v] += sum;
  }
}

__global__ void compute_bc_kernel(int num_nodes, int src_node, 
    int *d, float *delta, float *bc) {
  int v = blockIdx.x * blockDim.x + threadIdx.x;
  if (v >= num_nodes) {
    return;
  }

  if (src_node != v && d[v] != NOT_VISITED_MARKER) {
    bc[v] += delta[v];
  }
}

void setup(const graph *g, int **outgoing_starts, int **outgoing_edges, 
    int **d, int **sigma, int **distance,
    float **delta, float **bc, bool **done) {
  int num_nodes = g->num_nodes;
  int num_edges = g->num_edges;

  hipMalloc((void**)outgoing_starts, sizeof(int) * num_nodes);
  hipMalloc((void**)outgoing_edges, sizeof(int) * num_edges);
  hipMemcpy(outgoing_starts, g->outgoing_starts, sizeof(int) * num_nodes, hipMemcpyHostToDevice);
  hipMemcpy(outgoing_edges, g->outgoing_edges, sizeof(int) * num_edges, hipMemcpyHostToDevice);

  // TODO try using shared memory
  hipMalloc((void**)d, sizeof(int) * num_nodes);
  hipMalloc((void**)sigma, sizeof(int) * num_nodes);
  hipMalloc((void **)distance, sizeof(int));
  hipMalloc((void**)delta, sizeof(float) * num_nodes);
  hipMalloc((void**)bc, sizeof(float) * num_nodes);
  hipMalloc((void**)done, sizeof(bool));

  hipMemset(bc, 0, sizeof(float) * num_nodes);
}

void clean(int **d, int **sigma, int **distance, float **delta,
    float **bc, bool **done) {
  hipFree(d);
  hipFree(sigma);
  hipFree(distance);
  hipFree(delta);
  hipFree(bc);
  hipFree(done);
}

int gpu_bc_node (const graph *g, float *bc) {
  int *device_d, *device_sigma, *device_distance;
  int *device_outgoing_starts, *device_outgoing_edges;
  float *device_delta, *device_bc;
  bool *device_done;
  bool done;

#ifdef DEBUG
  double start_time = CycleTimer::currentSeconds();
#endif
  setup(g, &device_outgoing_starts, &device_outgoing_edges, &device_d, 
      &device_sigma, &device_distance, &device_delta, &device_bc, &device_done);

  dim3 blockDim(256);
  dim3 gridDim((g->num_nodes + blockDim.x - 1) / blockDim.x);

  for (int node_id = 0; node_id < g->num_nodes; ++node_id) {
    int distance = -1;

    // initialize parameters for d and sigma
    init_params<<<gridDim, blockDim>>>(g->num_nodes, device_d, device_sigma, node_id);
    
    // forward propagation
    do {
      done = true;
      ++distance;

#ifdef DEBUG
      std::cout << "forward, distance: " << distance << std::endl;
#endif

      hipMemset(device_done, true, sizeof(bool));
      hipMemcpy(device_distance, &distance, sizeof(int), hipMemcpyHostToDevice);

      forward_propagation_kernel<<<gridDim, blockDim>>>(device_outgoing_starts,
        device_outgoing_edges, g->num_nodes, g->num_edges, device_d, 
        device_sigma, device_distance, device_done);
      hipGetLastError();
      hipDeviceSynchronize();

      hipMemcpy(&done, device_done, sizeof(bool), hipMemcpyDeviceToHost);
    } while (!done);
#ifdef DEBUG
    std::cout << "node_id: " << node_id << ", distance: " << distance << std::endl;
#endif

    // backward propagation
    hipMemset(device_delta, 0, sizeof(float) * g->num_nodes);
    --distance;
    hipMemcpy(device_distance, &distance, sizeof(int), hipMemcpyHostToDevice);
    while (distance > 1) {
      backward_propagation_kernel<<<gridDim, blockDim>>>(device_outgoing_starts,
          device_outgoing_edges, g->num_nodes, g->num_edges, device_d, device_sigma, 
          device_delta, device_distance);
      --distance;
      hipMemcpy(device_distance, &distance, sizeof(int), hipMemcpyHostToDevice);
    }

    compute_bc_kernel<<<gridDim, blockDim>>>(g->num_nodes, node_id, device_d, 
        device_delta, device_bc);
  }

  hipMemcpy(bc, device_bc, sizeof(float) * g->num_nodes, hipMemcpyDeviceToHost);

  clean(&device_d, &device_sigma, &device_distance, &device_delta, 
      &device_bc, &device_done);

#ifdef DEBUG
  double total_time = CycleTimer::currentSeconds() - start_time;
  std::cout << "\ttotal time for gpu_bc_node: " << total_time << std::endl;
#endif

  return 0;
}
