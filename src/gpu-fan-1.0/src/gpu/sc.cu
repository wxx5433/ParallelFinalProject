#include "hip/hip_runtime.h"
/*
** Copyright (C) 2010 Zhiao Shi <zhiao.shi@accre.vanderbilt.edu>
**  
** This program is free software; you can redistribute it and/or modify
** it under the terms of the GNU General Public License as published by
** the Free Software Foundation; either version 2 of the License, or
** (at your option) any later version.
** 
** This program is distributed in the hope that it will be useful,
** but WITHOUT ANY WARRANTY; without even the implied warranty of
** MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
** GNU General Public License for more details.
** 
** You should have received a copy of the GNU General Public License
** along with this program; if not, write to the Free Software 
** Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA 02111-1307, USA.
*/

#include<iostream>
#include<unistd.h>
#include<cstdlib>
#include<string>
#include<cmath>
#include<algorithm>
#include<list>
#include"centrality.h"
#include"common.h"
#include"bitarray.h"

using namespace std;
extern int MAX_THREADS_PER_BLOCK;

__global__ void sc_bfs_kernel(int *d_v, int *d_e, int  *d_d, int *d_sigma,
  unsigned int *d_p, bool *d_continue, int *d_dist, int n_count, int e_count){
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if(tid<e_count){
    /* for each edge (u, w) */
    int u=d_v[tid];
    int w=d_e[tid];
    if(d_d[u]==*d_dist){
      if(d_d[w]==-1){
        *d_continue=true;
        d_d[w]=*d_dist+1;
      }
      if(d_d[w]==*d_dist+1){
        unsigned long long bit=(unsigned long long)w*n_count+u;
        atomicOr(d_p+BIT_INT(bit), (unsigned int) BIT_IN_INT(bit));
//        atomicAdd(&d_sigma[w],d_sigma[u]);
      }
    }
  }
}

__global__ void sc_bfs_set_init_sigma(int *d_dist, int *d_d, int *d_sigma, int n_count){
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if(tid<n_count){
    if(d_d[tid]==(*d_dist-1))
      d_sigma[tid]=1;
  }
}

__global__ void sc_bfs_back_prop_kernel(int *d_v, int *d_e, int *d_dist, int *d_d, int *d_sigma, unsigned int *d_p, 
     int n_count, int e_count){
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if(tid<e_count){
    /* for each edge (u, w) */
    int u=d_v[tid];
    int w=d_e[tid];
    if(d_d[u]==(*d_dist-1)){
      unsigned long long bit=(unsigned long long)u*n_count+w;
      if((d_p[BIT_INT(bit)] & BIT_IN_INT(bit)) != 0){
        atomicAdd(&d_sigma[w], d_sigma[u]);
      }
    }
  }                                    
}

__global__ void sc_bfs_back_sum_kernel(int s, int *d_dist, int *d_d, int *d_sc, int *d_sigma, int n_count){
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if(tid<n_count){
    if(tid!=s && d_d[tid]==(*d_dist-1))
      d_sc[tid]+=d_sigma[tid];
  }
}

/* stress centrality for unweighted graph 
 * intput: 
 *   n_count: the number of nodes
 *   e_count: the size of h_v and h_e
 *        for undirected graph, it is twice the number of edges
 *   h_v:  adjacency array 
 *   h_e:  adjacency array
 * output:
 *  h_sc: the stress centrality for each node
 *
 */
int sc_bfs(int n_count, int e_count, int *h_v, int *h_e, int *h_sc){
  int *d_v, *d_e;
  cudaCheckError(hipMalloc((void **)&d_v, sizeof(int)*e_count));
  cudaCheckError(hipMalloc((void **)&d_e, sizeof(int)*e_count));

  cudaCheckError(hipMemcpy(d_v, h_v, sizeof(int)*e_count, hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(d_e, h_e, sizeof(int)*e_count, hipMemcpyHostToDevice));

  int *d_d, *d_sigma;
  int *d_dist;
  int *d_sc;
  unsigned int *d_p; /* two dimensional bit array (nxn)*/

  cudaCheckError(hipMalloc((void **)&d_d, sizeof(int)*n_count));
  cudaCheckError(hipMalloc((void **)&d_sigma, sizeof(int)*n_count)); 
  cudaCheckError(hipMalloc((void **)&d_dist, sizeof(int)));
  cudaCheckError(hipMalloc((void **)&d_sc, sizeof(int)*n_count));
  unsigned long long total_bits=(unsigned long long)n_count*n_count;
  unsigned int num_of_ints=BITS_TO_INTS(total_bits);  
  cudaCheckError(hipMalloc((void **)&d_p, sizeof(unsigned int)*num_of_ints));
  cudaCheckError(hipMalloc((void **)&d_sc, sizeof(int)*n_count));
  cudaCheckError(hipMemset(d_sc, 0, sizeof(int)*n_count));

  int *h_d;
  h_d=(int *)malloc(sizeof(int)*n_count);
  for(int i=0; i<n_count; i++){
    for(int j=0; j<n_count; j++)
      h_d[j]=-1;
    h_d[i]=0;
    cudaCheckError(hipMemcpy(d_d, h_d, sizeof(int)*n_count, hipMemcpyHostToDevice));
    cudaCheckError(hipMemset(d_sigma, 0, sizeof(int)*n_count));
    cudaCheckError(hipMemset(d_p, 0, sizeof(unsigned int)*num_of_ints));     
    int threads_per_block=e_count;
    int blocks=1;
    if(e_count>MAX_THREADS_PER_BLOCK){
      blocks = (int)ceil(e_count/(float)MAX_THREADS_PER_BLOCK); 
      threads_per_block = MAX_THREADS_PER_BLOCK; 
    }
    dim3 grid(blocks);
    dim3 threads(threads_per_block);
    int threads_per_block2=n_count;
    int blocks2=1;
    if(n_count>MAX_THREADS_PER_BLOCK){
      blocks2 = (int)ceil(n_count/(double)MAX_THREADS_PER_BLOCK);
      threads_per_block2 = MAX_THREADS_PER_BLOCK; 
    }
    dim3 grid2(blocks2);
    dim3 threads2(threads_per_block2);
    bool h_continue;
    bool *d_continue;
    cudaCheckError(hipMalloc((void **)&d_continue, sizeof(bool)));
    int h_dist=0;
    cudaCheckError(hipMemset(d_dist, 0, sizeof(int)));
    // BFS  
    do{
      h_continue=false;
      cudaCheckError(hipMemcpy(d_continue, &h_continue, sizeof(bool), hipMemcpyHostToDevice));
      sc_bfs_kernel<<<grid,threads>>>(d_v, d_e, d_d, d_sigma, d_p, d_continue, d_dist, n_count, e_count);
      checkCUDAError("Kernel sc_bfs_kernel invocation");
      hipDeviceSynchronize();
      h_dist++; 
      cudaCheckError(hipMemcpy(d_dist, &h_dist, sizeof(int), hipMemcpyHostToDevice));
      cudaCheckError(hipMemcpy(&h_continue, d_continue, sizeof(bool), hipMemcpyDeviceToHost));
    }while(h_continue);   
    cudaCheckError(hipMemcpy(&h_dist, d_dist, sizeof(int), hipMemcpyDeviceToHost));
    sc_bfs_set_init_sigma<<<grid2,threads2>>>(d_dist, d_d, d_sigma, n_count);
    do{
      sc_bfs_back_prop_kernel<<<grid,threads>>>(d_v, d_e, d_dist, d_d, d_sigma, d_p, n_count, e_count);
      checkCUDAError("Kernel sc_bfs_back_prop_kernel invocation");
      hipDeviceSynchronize();
      h_dist--;
      cudaCheckError(hipMemcpy(d_dist, &h_dist, sizeof(int), hipMemcpyHostToDevice));
      sc_bfs_back_sum_kernel<<<grid2,threads2>>>(i, d_dist, d_d, d_sc, d_sigma, n_count);
      checkCUDAError("Kernel sc_bfs_back_sum_kernel invocation");
      hipDeviceSynchronize();
    }while(h_dist>1);
  }
  cudaCheckError(hipMemcpy(h_sc, d_sc, sizeof(int)*n_count, hipMemcpyDeviceToHost));
  free(h_d);
  hipFree(d_v);
  hipFree(d_e);
  hipFree(d_d);
  hipFree(d_sigma);
  hipFree(d_dist);
  hipFree(d_sc);
  return 0;
}

/* 
 * input: 
 *  h_v, h_e: the adjacency arrays
 *  output:
 *	 h_sc: the stress betweenness for each node
 */
int sc(bool directed, int n_count, int e_count, int * h_v, int *h_e,  int *h_sc){
  checkCUDADevice();
  sc_bfs(n_count, e_count, h_v, h_e, h_sc);
  /*
  if(!directed){
    for(int i=0; i<n_count; i++)
      h_sc[i]/=2.0;
  }
  */

  return 0;
}
