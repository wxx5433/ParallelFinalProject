#include"graph.h"
#include<stdio.h>
#include"common.h"

void checkCUDADevice(){
  int device_count;
  hipGetDeviceCount(&device_count);
  if(device_count<1){
    cout<<"No CUDA device."<<endl;
    exit(1);
  }
}

void checkCUDAError(const char *msg){
  hipError_t err = hipGetLastError();
  if(hipSuccess != err){
	 fprintf(stderr, "Cuda error: %s : %s.\n", msg, hipGetErrorString(err));
     exit(1);
  }    
}

void __cudaCheckError(hipError_t err, const char *file, int line ) { 
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
    exit( EXIT_FAILURE );
  }   
}
