#include "hip/hip_runtime.h"
/*
** Copyright (C) 2010 Zhiao Shi <zhiao.shi@accre.vanderbilt.edu>
**  
** This program is free software; you can redistribute it and/or modify
** it under the terms of the GNU General Public License as published by
** the Free Software Foundation; either version 2 of the License, or
** (at your option) any later version.
** 
** This program is distributed in the hope that it will be useful,
** but WITHOUT ANY WARRANTY; without even the implied warranty of
** MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
** GNU General Public License for more details.
** 
** You should have received a copy of the GNU General Public License
** along with this program; if not, write to the Free Software 
** Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA 02111-1307, USA.
*/

#include<iostream>
#include<unistd.h>
#include<cstdlib>
#include<string>
#include<cmath>
#include<algorithm>
#include"centrality.h"
#include"common.h"

using namespace std;

__global__ void cc_bfs_kernel(int *d_v, int *d_e, int  *d_d,
    bool *d_continue, int *d_dist, int e_count){
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if(tid<e_count){
    /* for each edge (u, w) */
    int u=d_v[tid];
    int w=d_e[tid];
    if(d_d[u]==*d_dist){
      if(d_d[w]==-1){
        *d_continue=true;
        d_d[w]=*d_dist+1;
      }   
    }   
  }
}

/* 
 * input: 
 *  v, e: the adjacency arrays
 *   source:  the source id 
 * output:
 *    res: the cost from the "source" to each node
 */
int cc_bfs(int n_count, int e_count, int *h_v, int *h_e, float *h_cc, bool ec){
  int *d_v, *d_e;


  cudaCheckError(hipMalloc((void **)&d_v, sizeof(int)*e_count));
  cudaCheckError(hipMalloc((void **)&d_e, sizeof(int)*e_count)); 

  cudaCheckError(hipMemcpy(d_v, h_v, sizeof(int)*e_count, hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(d_e, h_e, sizeof(int)*e_count, hipMemcpyHostToDevice));

  int *d_d, *d_dist; 

  cudaCheckError(hipMalloc((void **)&d_d, sizeof(int)*n_count));
  cudaCheckError(hipMalloc((void **)&d_dist, sizeof(int)));

  int *h_d;
  h_d=(int *)malloc(sizeof(int)*n_count);
  bool *d_continue;
  cudaCheckError(hipMalloc((void**)&d_continue, sizeof(bool)));
  
  for(int s=0; s<n_count; s++){
    for(int i=0; i<n_count; i++)
      h_d[i]=-1;
    h_d[s]=0;
    cudaCheckError(hipMemcpy(d_d, h_d, sizeof(int)*n_count, hipMemcpyHostToDevice));
    int threads_per_block=e_count;
    int blocks=1;
    if(e_count>MAX_THREADS_PER_BLOCK){
      blocks = (int)ceil(e_count/(float)MAX_THREADS_PER_BLOCK); 
      threads_per_block = MAX_THREADS_PER_BLOCK; 
    }
    dim3 grid(blocks);
    dim3 threads(threads_per_block);
    bool h_continue;
    int h_dist=0;
    cudaCheckError(hipMemset(d_dist, 0, sizeof(int)));
    do{
      h_continue=false;
      cudaCheckError(hipMemcpy(d_continue, &h_continue, sizeof(bool), hipMemcpyHostToDevice));
      cc_bfs_kernel<<<grid, threads>>>(d_v, d_e, d_d, d_continue, d_dist, e_count);
      checkCUDAError("Kernel invocation");
      hipDeviceSynchronize();
      h_dist++;
      cudaCheckError(hipMemcpy(d_dist, &h_dist, sizeof(int), hipMemcpyHostToDevice));
      cudaCheckError(hipMemcpy(&h_continue, d_continue, sizeof(bool), hipMemcpyDeviceToHost));
    }while(h_continue);
    cudaCheckError(hipMemcpy(h_d, d_d, sizeof(int)*n_count, hipMemcpyDeviceToHost));
    if(!ec){
      int sum=0;
      int connected=n_count;
      for(int i=0; i<n_count; i++){
        if(h_d[i]==-1) 
          connected--; 
        else
          sum+=h_d[i];
      }
      if(connected==0) //isolated
        h_cc[s]=0;
      else
        h_cc[s]=1.0*(connected-1)*(connected-1)/(n_count-1)/sum;
    }
    else{ //eccentricity
      int max=-1;
      int connected=n_count;
      for(int i=0; i<n_count; i++){
        if(h_d[i]==-1) 
          connected--; 
        else{
          if(h_d[i]>max)
            max=h_d[i];
        }
      }
      if(connected==0)
        h_cc[s]=0;
      else
        h_cc[s]=1.0*(connected-1)*(connected-1)/(n_count-1)/max;
    }
  }
  free(h_d);
  hipFree(d_d);
  hipFree(d_continue);
  hipFree(d_v);
  hipFree(d_e);
  hipFree(d_dist);
  return 0;
}

/* if ec==true, compute eccentricity centrality */
int cc(int n_count, int e_count, int * h_v, int *h_e,  float *h_c, bool ec){
  checkCUDADevice();
  cc_bfs(n_count, e_count, h_v, h_e, h_c, ec);
  return 0;
}                                 
